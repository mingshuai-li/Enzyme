#include "hip/hip_runtime.h"
#include <stdio.h>

void __device__ square_impl(double* x_in, double *x_out) {
    x_out[0] = x_in[0] * x_in[0];
}

typedef void (*f_ptr)(double*, double*);

extern void __device__ __enzyme_autodiff(f_ptr,
    int, double*, double*,
    int, double*, double*
);

void __global__ square(double* x_in, double *x_out) {
    square_impl(x_in, x_out);
}

int __device__ enzyme_dup;
int __device__ enzyme_out;
int __device__ enzyme_const;

void __global__ square_grad(double* x, double *d_x, double *y, double *d_y) {

    __enzyme_autodiff(square_impl,
        enzyme_dup, x, d_x,
        enzyme_dup, y, d_y);

}

int main() {

    // Device pointers
    double *x, *d_x, *y, *d_y;

    // Allocate GPU device memory
    hipMalloc(&x, sizeof(*x));
    hipMalloc(&d_x, sizeof(*d_x))
    hipMalloc(&y, sizeof(*y));
    hipMalloc(&d_y, sizeof(*d_y))

    // Initialize device values
    double host_x = 1.4;
    double host_d_x = 0.0;
    double host_y;
    double host_d_y = 1.0;

    // Copy data to device
    hipMemcpy(x, &host_x, sizeof(*x), hipMemcpyHostToDevice);
    hipMemcpy(d_x, &host_d_x, sizeof(*d_x), hipMemcpyHostToDevice);
    hipMemcpy(y, &host_y, sizeof(*y), hipMemcpyHostToDevice);
    hipMemcpy(d_y, &host_d_y, sizeof(*d_y), hipMemcpyHostToDevice);

#ifdef FORWARD
    // Forward pass only
    square<<<1, 1>>>(x, y);
#else
    // Forward and backward pass
    square_grad<<<1, 1>>>(x, d_x, y, d_y);  
#endif

    // Synchronize device memory
    hipDeviceSynchronize();

    // Copy data from device to host
    hipMemcpy(&host_x, x, sizeof(*x), hipMemcpyDeviceToHost);
    hipMemcpy(&host_d_x, d_x, sizeof(*d_x), hipMemcpyDeviceToHost);
    hipMemcpy(&host_y, y, sizeof(*y), hipMemcpyDeviceToHost);
    hipMemcpy(&host_d_y, d_y, sizeof(*d_y), hipMemcpyDeviceToHost);

    // Print results
    printf("%f %f\n", host_x, host_y);
    printf("%f %f\n", host_d_x, host_d_y);
    
}